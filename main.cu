// Toy program to exercise gpu offloading.
// https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/
//
// CUDA functions:
//  cudaMalloc(void **devPtr, size_t count);
//  cudaFree(void *devPtr);
//  cudaMemcpy(void *dst, void *src, size_t count, cudaMemcpyKind kind)
//  cudaMemcpyHostToDevice or cudaMemcpyDeviceToHost

#include <assert.h>
#include <math.h>
#include <stdio.h>

#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a = (float*)malloc(sizeof(*a) * N);
    float *b = (float*)malloc(sizeof(*b) * N);
    float *out = (float*)malloc(sizeof(*out) * N);

    for (int i = 0; i < N; ++i) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }
    memset(out, 0, sizeof(*out) * N);

    float *gpu_a;
    float *gpu_b;
    float *gpu_out;
    hipMalloc((void**)&gpu_a, sizeof(*gpu_a) * N);
    hipMalloc((void**)&gpu_b, sizeof(*gpu_b) * N);
    hipMalloc((void**)&gpu_out, sizeof(*gpu_out) * N);

    hipMemcpy(gpu_a, a, sizeof(*a) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, sizeof(*b) * N, hipMemcpyHostToDevice);

    vector_add<<<1,1>>>(gpu_out, gpu_a, gpu_b, N);

    hipMemcpy(out, gpu_out, sizeof(*out) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        if (fabs(out[i] - a[i] - b[i]) >= MAX_ERR) {
            fprintf(
                stderr,
                "out[%d]=%f - a[%d]=%f - b[%d]=%f > %f\n",
                i,
                out[i],
                i,
                a[i],
                i,
                b[i],
                MAX_ERR);
            assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
        }
    }
    fprintf(stderr, "out[0] = %f\n", out[0]);
    fprintf(stderr, "PASSED\n");

    free(a);
    free(b);
    free(out);

    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_out);

    return 0;
}
